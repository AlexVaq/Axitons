#include "hip/hip_runtime.h"
#include "enum-vars.h"
#include "cudaErrors.h"

#define	TwoPi	(2.*M_PI)

template<typename Float>
static __device__ __forceinline__ Float modPi (const Float x, const Float OneOvPi, const Float TwoPiZ)
{
	const Float tmp = x*OneOvPi;

	if (tmp >=  1.)
		return (x-TwoPiZ);

	if (tmp <  -1.)
		return (x+TwoPiZ);

	return x;
}

template<typename Float, const int nNeig>
static __device__ __forceinline__ constexpr Float C(int nIdx) {

	switch (nNeig) {

		default:
		case 1: {
			return	((Float) 1.0);
			break;
		}

		case 2: {
			constexpr Float C[2] = { 4.0/3.0, -1.0/12.0 };
			return	C[nIdx];
			break;
		}

		case 3: {
			constexpr Float C[3] = { 1.5,     -3.0/20.0,  1.0/90.0 };
			return	C[nIdx];
			break;
		}

		case 4: {
			constexpr Float C[4] = { 8.0/5.0, -1.0/5.0,   8.0/315.0, -1.0/560.0 };
			return	C[nIdx];
			break;
		}
	}
}


template<typename Float, const int nNeig, const bool wMod>
static __device__ __forceinline__ void	propagateCoreGpu(const uint idx, const Float * __restrict__ field, Float * __restrict__ dev, Float * __restrict__ misc, const Float gm, const Float zQ,
							 const Float iz, const Float dzc, const Float dzd, const Float ood2, const int Lx, const Float zP, const Float tPz)
{
	Float mel = 0.0, pPc, a, f0n;
	f0n = field[idx];

	if (idx != 0) {
		pPc = 1.0/((Float) idx);
		if (idx < Lx - nNeig){
			#pragma unroll
			for (int nIdx=1; nIdx<=nNeig; nIdx++)
			{
				auto rIdx  = __sad (idx, nIdx, 0);
				// mel       += (field[idx+nIdx]*(1.0 + nIdx*pPc) + field[rIdx]*((Float) rIdx)*pPc - 2.0*f0n)*C<Float,nNeig>(nIdx-1);
				mel       += (field[idx+nIdx]*(1.0 + nIdx*pPc) + field[rIdx]*(1.0 - nIdx*pPc) - 2.0*f0n)*C<Float,nNeig>(nIdx-1);
			}
		} else {
			// mel += (field[idx-2]-f0n)*(1.0 + pPc) - 2.0*(field[idx-1]-f0n)*(1 + 2.0*pPc);
			mel +=  -(field[idx-3]-f0n)*(1.0 + 2.0*pPc/3.0)  + (field[idx-2]-f0n)*(4.0 + 3.0*pPc) - (field[idx-1]-f0n)*(5.0 + 6.0*pPc);
		}
	} else {
		#pragma unroll
		for (int nIdx=1; nIdx<=nNeig; nIdx++) {
			mel += (field[nIdx] - f0n)*2.0*C<Float,nNeig>(nIdx-1);
		}
	}

	// if (idx != 0) { // FIXME SLOW AS HELL
	// 	pPc = 1.0/((Float) idx);
	// 	#pragma unroll
	// 	for (int nIdx=1; nIdx<=nNeig; nIdx++)
	// 	{
	// 		auto rIdx  = __sad (idx, nIdx, 0);
	// 		mel       += (field[idx+nIdx]*(1.0 + nIdx*pPc) + field[rIdx]*((Float) rIdx)*pPc - 2.0*f0n)*C<Float,nNeig>(nIdx-1);
	// 	}
	// } else {
	// 	#pragma unroll
	// 	for (int nIdx=1; nIdx<=nNeig; nIdx++) {
	// 		mel += (field[nIdx] - f0n)*2.0*C<Float,nNeig>(nIdx-1);
	// 	}
	// }


/*
	if (idx > nNeig) { // FIXME SLOW AS HELL
		pPc = 1.0/cIdx;
		#pragma unroll
		for (int nIdx=1; nIdx<nNeig; nIdx++)
			mel += (field[idx+nIdx]*(((Float) nIdx) + pPc) + field[idx-nIdx]*(((Float) nIdx) - pPc))*C[nIdx] - f0n*C[nIdx];
	} else {
		for (int cIdx=0;     i<idx;   i++)
			mel += (field[cIdx+idx]*(((Float) nIdx) + pPc) + field[cIdx-idx]*(((Float) nIdx) - pPc))*C[cIdx] - f0n*C[cIdx];

		mel += (field[idx*2]*(((Float) idx) + pPc) + f0n*(((Float) idx) - pPc - 1))*C[idx];

		for (int cIdx=idx+1; i<nNeig; i++)
			mel += (field[idx+cIdx]*(((Float) cIdx) + pPc) + field[idx-cIdx]*(((Float) cIdx) - pPc))*C[cIdx] - f0n*C[cIdx];
	}
*/
	a = mel*ood2 - zQ*sin(f0n*iz);

	mel	 = dev[idx];

	if ((idx > Lx*0.8) && (gm > 0.0) ) {	// FIXME No hardcode this 0.8
		// variable measured with respect to axion mass?
		Float nGm = gm*(idx-0.8*Lx)/((Float) idx)*5;
		a         = (a - nGm*mel)/(1. + 0.5*nGm*dzc);
	}

	mel	+= a*dzc;
	dev[idx] = mel;
	mel	*= dzd;
	f0n	+= mel;

	misc[idx] = f0n; //modPi(tmp, zP, tPz);
}

template<typename Float, const int nNeig, const bool wMod>
__global__ void	propagateKernel(const Float * __restrict__ field, Float * __restrict__ dev, Float * __restrict__ misc, const Float gamma, const Float zQ, const Float dzc, const Float dzd,
				const Float ood2, const Float iz, const int Lx, const Float zP=0, const Float tPz=0)
{
	uint idx = threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y);
	//uint idx = Vo + (threadIdx.x + blockDim.x*blockIdx.x) + Sf*(threadIdx.y + blockDim.y*blockIdx.y);

	// if	(idx >= Lx - nNeig)
	// 	return;
	if	(idx < Lx)
		propagateCoreGpu<Float,nNeig,wMod>(idx, field, dev, misc, gamma, zQ, iz, dzc, dzd, ood2, Lx, zP, tPz);
}

void	propGpu(const void * __restrict__ field, void * __restrict__ dev, void * __restrict__ misc, const double z, const double dz, const double c, const double d, const double ood2,
		const double aMass2, const int Lx, FieldPrecision precision, const int nNeig, const double gamma, const int xBlock, const int yBlock, const int zBlock)
{
	#define	BLSIZE 256
	dim3 gridSize((Lx+BLSIZE-1)/BLSIZE,1,1);
	dim3 blockSize(BLSIZE,1,1);

	if (precision == DoublePrecision)
	{
		const double dzc   = dz*c;
		const double dzd   = dz*d;
		const double zQ    = aMass2*z*z*z;//axionmass2((double) zR, nQcd, zthres, zrestore)*zR*zR*zR;
		const double iZ    = 1./z;
		const double gdzc2 = gamma*sqrt(aMass2)*z;

		switch (nNeig) {
			default:
			case 1:
			propagateKernel<double,1,false><<<gridSize,blockSize,0>>>((const double *) field, (double *) dev, (double *) misc, gdzc2, zQ, dzc, dzd, ood2, iZ, Lx);
			break;

			case 2:
			propagateKernel<double,2,false><<<gridSize,blockSize,0>>>((const double *) field, (double *) dev, (double *) misc, gdzc2, zQ, dzc, dzd, ood2, iZ, Lx);
			break;

			case 3:
			propagateKernel<double,3,false><<<gridSize,blockSize,0>>>((const double *) field, (double *) dev, (double *) misc, gdzc2, zQ, dzc, dzd, ood2, iZ, Lx);
			break;

			case 4:
			propagateKernel<double,4,false><<<gridSize,blockSize,0>>>((const double *) field, (double *) dev, (double *) misc, gdzc2, zQ, dzc, dzd, ood2, iZ, Lx);
			break;
		}
	}
	else if (precision == SinglePrecision)
	{
		const float dzc   = dz*c;
		const float dzd   = dz*d;
		const float zQ    = (float) (aMass2*z*z*z);//axionmass2((double) zR, nQcd, zthres, zrestore)*zR*zR*zR;
		const float iZ    = 1./z;
		const float gdzc2 = (float) gamma*sqrt(aMass2)*z;

		switch (nNeig) {
			default:
			case 1:
			propagateKernel<float, 1,false><<<gridSize,blockSize,0>>>((const float *) field, (float *) dev, (float *) misc, gdzc2, zQ, dzc, dzd, (float) ood2, iZ, Lx);
			break;

			case 2:
			propagateKernel<float, 2,false><<<gridSize,blockSize,0>>>((const float *) field, (float *) dev, (float *) misc, gdzc2, zQ, dzc, dzd, (float) ood2, iZ, Lx);
			break;

			case 3:
			propagateKernel<float, 3,false><<<gridSize,blockSize,0>>>((const float *) field, (float *) dev, (float *) misc, gdzc2, zQ, dzc, dzd, (float) ood2, iZ, Lx);
			break;

			case 4:
			propagateKernel<float, 4,false><<<gridSize,blockSize,0>>>((const float *) field, (float *) dev, (float *) misc, gdzc2, zQ, dzc, dzd, (float) ood2, iZ, Lx);
			break;
		}
	}
}
/*
void	propModGpu(const void * __restrict__ field, void * __restrict__ dev, void * __restrict__ misc, const double z, const double dz, const double c, const double d, const double ood2,
		   const double aMass2, const int Lx, FieldPrecision precision, const int xBlock, const int yBlock, const int zBlock)
{
	dim3 gridSize((Lx+BLSIZE-1)/BLSIZE,1,1);
	dim3 blockSize(BLSIZE,1,1);
	//dim3 gridSize((Sf+xBlock-1)/xBlock,(Lz2+yBlock-1)/yBlock,1);
	//dim3 blockSize(xBlock,yBlock,1);

	if (precision == DoublePrecision)
	{
		const double dzc  = dz*c;
		const double dzd  = dz*d;
		const double zQ   = aMass2*z*z*z;//xionmass2((double) zR, nQcd, zthres, zrestore)*zR*zR*zR;
		const double iZ   = 1./z;
		const double tPz  = 2.*M_PI*z;
		propagateKernel<double,true><<<gridSize,blockSize,0>>>((const double*) field, (double*) dev, (double*) misc, zQ, dzc, dzd, ood2, iZ, Lx, M_1_PI*iZ, tPz);
	}
	else if (precision == SinglePrecision)
	{
		const float dzc = dz*c;
		const float dzd = dz*d;
		const float zQ = (float) (aMass2*z*z*z);//axionmass2((double) zR, nQcd, zthres, zrestore)*zR*zR*zR;
		const float iZ   = 1./z;
		const float tPz  = 2.*M_PI*z;
		propagateKernel<float, true><<<gridSize,blockSize,0>>>((const float *) field, (float *) dev, (float *) misc, zQ, dzc, dzd, ood2, iZ, Lx, M_1_PI*iZ, tPz);
	}
}
*/
void	propGpu(const void * __restrict__ field, void * __restrict__ dev, void * __restrict__ misc, const double z, const double dz, const double c, const double d, const double ood2,
		const double aMass2, const int Lx, FieldPrecision precision, const int nNeig, const double gamma, const int xBlock, const int yBlock, const int zBlock, const FieldType wMod)
{
	switch (wMod) {

		case	FieldCompact:
			printf ("Compact propagator not implemented\n");
			//propModGpu(field, dev, misc, z, dz, c, d, ood2, aMass2, Lx, precision, gamma, xBlock, yBlock, zBlock);
			break;

		case	FieldNonCompact:
			propGpu	  (field, dev, misc, z, dz, c, d, ood2, aMass2, Lx, precision, nNeig, gamma, xBlock, yBlock, zBlock);
			break;
	}

	CudaCheckError();

	return;
}
